/* 
 * File:   MBSet.cu
 * 
 * Created on June 24, 2012
 * 
 * Purpose:  This program displays Mandelbrot set using the GPU via CUDA and
 * OpenGL immediate mode.
 * 
 */

#include <iostream>
#include <stack>
#include <hip/hip_runtime_api.h>
#include <stdio.h>
#include "Complex.cu"

#include <GL/freeglut.h>

// Size of window in pixels, both width and height
#define WINDOW_DIM            (512)
// Maximum iterations for the MBSet calculations
#define MAX_IT                (2000)

using namespace std;

// Initial screen coordinates, both host and device.
Complex minC(-2.0, -1.2);
Complex maxC(1.0, 1.8);
Complex* dev_minC;
Complex* dev_maxC;

// Block and thread counts for CUDA
dim3 blocks(WINDOW_DIM/8, WINDOW_DIM/8);
dim3 threads(8, 8);



int main(int argc, char** argv)
{
  // Initialize OPENGL here
  // Set up necessary host and device buffers
  // set up the opengl callbacks for display, mouse and keyboard

  // Calculate the interation counts
  // Grad students, pick the colors for the 0 .. 1999 iteration count pixels

  glutMainLoop(); // THis will callback the display, keyboard and mouse
  return 0;
  
}
