
#include <hip/hip_runtime.h>
/* 
 * File:   Complex.cu
 * Simple complex number class for use with CUDA
 */

class Complex 
{
public:
    float   r;
    float   i;
    __host__ __device__ Complex( float a, float b ) : r(a), i(b)  {}
    __device__ Complex(const Complex& x) : r(x.r), i(x.i) {}
    __device__ float magnitude2( void ) {
        return r * r + i * i;
    }
    __device__ Complex operator*(const Complex& a) {
        return Complex(r*a.r - i*a.i, i*a.r + r*a.i);
    }
    __device__ Complex operator+(const Complex& a) {
        return Complex(r+a.r, i+a.i);
    }
};